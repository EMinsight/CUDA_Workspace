#define _USE_MATH_DEFINES
#include <iostream>
#include <cmath>
#include <fstream>
#include <string>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "main.h"

constepr int Num_Individual { 1 };

int main( void ){

    std::ofstream ofs;

    ofs.open("./result/Magnitude.dat");

    perturbation *P_info = new perturbation[Num_Individual];
    for( int i = 0; i < Num_Individual; i++ ){
        P_info[i].set_alpha( 10.0 );
        P_info[i].set_center( 74.0, Nr/2, Nphi/2 );
        P_info[i].set_sigma( 2.0e3, 60.0e3 );
    }

    /* Set Y(Year)M(Month)D(Date) */
    date ymd;
    ymd.set_ymd( 2016, 3, 1 );
    ymd.set_h( 9.0 );

    /* Set geocoordinate points */
    geocoordinate lla_info;
    lla_info.set_point( 32.0, 135.0, (Alt_lower_ionosphere/1.0e3) );

    /* Observation Points on propagation path */
    int Num_obs = ( Nphi -2*L ) - k_s + 1;

    geocoordinate *obs_p = new geocoordinate[Num_obs];
    for( int k = 0; k < Num_obs; k++ ){
        obs_p[k].set_obs( 0, 50, k + k_s );
    }

    /* Magnitude */
    float *Magnitude = new float[ Num_Individual*Num_obs ];

    return 0;
}