#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES
#include <iostream>
#include <cmath>
#include <complex>
#include <fstream>
#include <string>
#include <chrono>

#include "fdtd3d.h"

const float R_r{ 100.0e3 };

const float delta_r = R_r/float(Nr);
const float delta_theta = 1.0e3/float(R0);
const float delta_phi = 1.0e3/float(R0);
const float Dt = float( 0.99/C0/std::sqrt(1.0/delta_r/delta_r
 + 1.0/R0/R0/delta_theta/delta_theta
 + 1.0/R0/R0/std::sin(THETA0)/std::sin(THETA0)/delta_phi/delta_phi) );
const float inv_Dt = 1.0/Dt;
const float sigma_t = 7*Dt;
const float t0 = 6.0*sigma_t;

// PML info //
const int L { 10 };
const float M {3.5 };
const float R { 1.0e-6 };

const float sigma_th_max = float( -(M + 1.0)*C0*std::log(R)/2.0/double(L)/delta_theta/R0 );
const float sigma_phi_max = float( -(M + 1.0)*C0*std::log(R)/2.0/double(L)/delta_phi/R0 );

// Ionosphere info //
constexpr float Alt_lower_ionosphere { 60.0e3 };
const int ion_L = int( (R_r - Alt_lower_ionosphere )/delta_r );
const float freq { 22.2e3 };
const float omega = 2.0*M_PI*freq;

// Geomagnetic info //
const float B_abs = float( 4.6468e-5 );
const float Dec = float( -7.0*M_PI/180.0 );
const float Inc = float( 49.0*M_PI/180.0 );
const float Azim = float( 61.0*M_PI/180.0 );

void fdtd_calc(perturbation P_info, date ymd, geocoordinate lla_info, 
                            int Num_obs, geocoordinate* obs_p, float* Magnitude )
{
    int t_step = 1800;
    float t;
    float J;
    int NEW;
    int OLD;
    std::complex <float> zj(0.0, 1.0);

    float* Hr, *Htheta, *Hphi;
    Hr = new float [ (Nr + 1)*Ntheta*Nphi ];
    Htheta = new float [ Nr*(Ntheta + 1)*Nphi ];
    Hphi = new float [ Nr*Ntheta*(Nphi + 1) ];
    array_initialize( Hr, (Nr + 1)*Ntheta*(Nphi) );
    array_initialize( Htheta, Nr*(Ntheta + 1)*Nphi );
    array_initialize( Hphi, Nr*Ntheta*(Nphi + 1) )

    float* Er, *Etheta, *Ephi;
    Er = new float[ 2*Nr*(Ntheta + 1)*(Nphi + 1)　];
    Etheta = new float[ 2*(Nr + 1)*Ntheta*(Nphi + 1) ];
    Ephi = new float[ 2*(Nr + 1)*(Ntheta + 1)*Nphi ];
    array_initialize( Er, 2*Nr*(Ntheta + 1)*(Nphi + 1) );
    array_initialize( Etheta, 2*(Nr + 1)*Ntheta*(Nphi + 1) );
    array_initialize( Ephi, 2*(Nr + 1)*(Ntheta + 1)*Nphi );

    float* Dr, *Dtheta, *Dphi;
    Dr = new float[ 2*Nr*(Ntheta + 1)*(Nphi + 1)　];
    Dtheta = new float[ 2*(Nr + 1)*Ntheta*(Nphi + 1) ];
    Dphi = new float[ 2*(Nr + 1)*(Ntheta + 1)*Nphi ];
    array_initialize( Dr, 2*Nr*(Ntheta + 1)*(Nphi + 1) );
    array_initialize( Dtheta, 2*(Nr + 1)*Ntheta*(Nphi + 1) );
    array_initialize( Dphi, 2*(Nr + 1)*(Ntheta + 1)*Nphi );

    float *Dr_theta1, *Dr_theta2, *Dr_phi;
    float *Dtheta_phi, *Dtheta_r;
    float *Dphi_r, *Dphi_theta;

    float *Hr_theta1, *Hr_theta2, *Hr_phi;
    float *Htheta_phi, *Htheta_r;
    float *Hphi_r, *Hphi_theta;

    /////////////////////////////////////////////////
    ////////////////ここをどうするか/////////////////
    /////////////////////////////////////////////////

    pml* idx_Dr = new pml[4];
    pml* idx_Dth = new pml[4];
    pml* idx_Dphi = new pml[4];
    pml* idx_Hr = new pml[4];
    pml* idx_Hth = new pml[4];
    pml* idx_Hphi = new pml[4];

    pml_idx_initialize(
        idx_Dr, idx_Dth, idx_Dphi,
        idx_Hr, idx_Hth, idx_Hphi
    );

    float *sigma_theta, *sigma_phi, *sigma_thetea_h, *sigma_phi_h;
    sigma_theta = new float[ Ntheta + 1 ];
    sigma_phi = new float[ Nphi + 1 ];
    sigma_theta_h = new float[ Ntheta + 1 ];
    sigma_phi_h = new float[ Nphi + 1 ];

    float *geo_B = new float[3];
    float *sph_B = new float[3];

    geo_mag( geo_B, sph_B );

    // Ne, nyu //
    float *Nh = new float[ion_L];
    //float *noise_Nh = new float[(ion_L + 1)*(Ntheta + 1)*(Nphi + 1)];
    float *noise_Nh = new float[ion_L*Ntheta*Nphi];
    float *ny = new float[ion_L + 1];
    float *Re = new float[ion_L + 1]; 

    Ne_allocate(Nh, Re);
    ny_allocate(ymd, lla_info, ny, Re);

    float *Cmat = new float[ion_L*Ntheta*Nphi*3*3];
    float *Fmat = new float[ion_L*Ntheta*Nphi*3*3];

    set_perturbation( P_info, noise_Nh, Nh );
    set_matrix( zj, Cmat, Fmat, noise_Nh, ny );

    // calculate surface impedance //
    std::complex <float> Z(0.0, 0.0);
    float Z_real, Z_imag;

    Z = surface_impe(zj);

    Z_real = Z.real();
    Z_imag = Z.imag()/omega;

    t = Dt*0.0;

    // fourie //
    std::complex <float>* E_famp = new std::complex <float> [Num_obs];

}