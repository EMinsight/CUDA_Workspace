#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES
#include <cmath>
#include "main.h"

__global__ void update_Hx( int Nx, int Ny, float* Hx_d, float* Ez_d, float CHx )
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;

    if( (i > 0) && ( i < Nx) && ( j >= 0) && ( j < Ny) ){
        Hx_d[i + j*(Nx+1)] = Hx_d[i + j*(Nx+1)]
         - CHx * ( Ez_d[i + (j+1)*(Nx+1)] - Ez_d[i + j*(Nx+1)] );
    }

}