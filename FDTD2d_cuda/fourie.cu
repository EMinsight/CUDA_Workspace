#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES
#include <cmath>
#include <complex>
#include "main.h"

__global__ void fourie( int Nx, int Ny, std::complex <float> zj, float omega,
                        float t, float *Ez_d, std::complex <float>* Ez_famp_d )
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;

    if( j == (Ny/2) ){
        Ez_famp_d[i] = Ez_famp_d[i]
                 + Ez_d[i + j*(Nx + 1)]*std::exp( -zj*omega*t )*Dt;
    }
}