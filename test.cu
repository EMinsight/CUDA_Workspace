#include <iostream>
#include <cmath>
#include <fstream>
#include <string>
#include <stdio.h>
#include <hip/hip_runtime.h>


constexpr float C0 { 3.0e8 };
constexpr float MU0 { 4.0e-7 * M_PI };
constexpr float EPS0 { 1.0/MU0/C0/C0 };

constexpr int Nx { 100 };
constexpr int Ny { 100 };
constexpr float Dx { 1.0e3 };
constexpr float Dy { 1.0e3 };
constexpr float Tmax { .5e-3 };

int idx_ez(int i, int j){
  return i*(Ny+1) + j;
  //return i + j*(Nx+1);
}

int idx_hx(int i, int j){
  return i*Ny + j;
  //return i + j*(Nx + 1);
}

int idx_hy(int i, int j){
  return i*(Ny+1) + j;
  //return i + j*Nx;
}

__global__ void update_Ez(float *Ez_d, float *Hx_d, float *Hy_d,
int Nx, int Ny, float CEz1, float CEz2){

  int i = blockDim.x * blockIdx.x + threadIdx.x;
  int j = blockDim.y * blockIdx.y + threadIdx.y;

  if ( (i > 0) && (i < Nx) && (j > 0) && (j < Ny) ){
    Ez_d[i*(Ny+1)+j] = Ez_d[i*(Ny+1)+j]
    + CEz1 * (Hy_d[i*(Ny+1) + j] - Hy_d[(i-1)*(Ny+1) + j])
    - CEz2 * (Hx_d[i*Ny + j]     - Hx_d[i*Ny + j-1]);
  }
  /*if( (i > 0) && (i < Nx) && (j > 0) && (j < Ny)){
        Ez_d[i + j*(Nx + 1)] = Ez_d[i + j*(Nx + 1)]
                            + CEz1 * (Hy_d[i + j*Nx] - Hy_d[(i-1) + j*Nx])
                            - CEz2 * (Hx_d[i + j*(Nx + 1)] - Hx_d[i + (j-1)*(Nx + 1)]);
  }*/
}

__global__ void update_Hx(float *Hx_d, float *Ez_d,
int Nx, int Ny, float CHx){
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  int j = blockDim.y * blockIdx.y + threadIdx.y;

  if ( (i > 0) && (i < Nx) && (j >= 0) && (j < Ny) ){
    Hx_d[i*Ny + j] = Hx_d[i*Ny + j]
    - CHx * (Ez_d[i*(Ny+1) + j+1] - Ez_d[i*(Ny+1) + j]);
  }
  /*if( (i > 0) && ( i < Nx) && ( j >= 0) && ( j < Ny) ){
        Hx_d[i + j*(Nx+1)] = Hx_d[i + j*(Nx+1)]
         - CHx * ( Ez_d[i + (j+1)*(Nx+1)] - Ez_d[i + j*(Nx+1)] );
  }*/
}

__global__ void update_Hy(float *Hy_d, float *Ez_d,
int Nx, int Ny, float CHy){
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  int j = blockDim.y * blockIdx.y + threadIdx.y;

  if ( (i >= 0) && (i < Nx) && (j > 0) && (j < Ny) ){
    Hy_d[i*(Ny+1) + j] = Hy_d[i*(Ny+1) + j]
    + CHy * (Ez_d[(i+1)*(Ny+1) + j] - Ez_d[i*(Ny+1) + j]);
  }
  /*if( (i >= 0) && (i < Nx) && (j > 0) && (j < Ny) ){
        Hy_d[i + j*Nx] = Hy_d[i + j*Nx]
                + CHy*( Ez_d[(i + 1) + j*(Nx+1)] - Ez_d[i + j*(Nx+1)] );
  }*/
}

__global__ void add_Jz(float *Ez_d, int i_s, int j_s,
float t, float Dt, float EPS0, float sig, float t0){
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  int j = blockDim.y * blockIdx.y + threadIdx.y;

  if ( (i == i_s) && (j == j_s) ){
    Ez_d[i*(Ny+1)+j] = Ez_d[i*(Ny+1)+j]
    - Dt/EPS0 * exp( - (t - t0)*(t - t0) / 2.0 / sig/sig );

  }
  /*if( (i == i_s) && (j == j_s) ){
        Ez_d[i + j*(Nx + 1)] = Ez_d[i + j*(Nx + 1)]
         - Dt/EPS0 * std::exp( -(t - t0)*(t - t0) / 2.0 / sig / sig );
  }*/
}

int main(void){
  const float Dt { float(0.999/C0/std::sqrt(1./Dx/Dx + 1./Dy/Dy)) };
  const int NT { int(Tmax/Dt) };
  std::cout << Dt << ", " << NT << std::endl;

  const float sig { 10*Dt };

  const float CEz1 { Dt/EPS0/Dx };
  const float CEz2 { Dt/EPS0/Dy };
  const float CHx { Dt/MU0/Dy };
  const float CHy { Dt/MU0/Dx };

  float *Ez = new float [ (Nx+1)*(Ny+1) ];
  float *Hx = new float [ (Nx+1)*Ny     ];
  float *Hy = new float [ Nx    *(Ny+1) ];
  for(int i = 0; i <= Nx; i++){
    for(int j = 0; j <= Ny; j++){
      Ez[idx_ez(i,j)] = 0.0;
    }
  }
  for(int i = 0; i <= Nx; i++){
    for(int j = 0; j < Ny; j++){
      Hx[idx_hx(i,j)] = 0.0;
    }
  }
  for(int i = 0; i < Nx; i++){
    for(int j = 0; j <= Ny; j++){
      Hy[idx_hy(i,j)] = 0.0;
    }
  }


  float *Ez_d, *Hx_d, *Hy_d;
  hipMalloc( (void**)&Ez_d, sizeof(float)*(Nx+1)*(Ny+1) );
  hipMalloc( (void**)&Hx_d, sizeof(float)*(Nx+1)*Ny );
  hipMalloc( (void**)&Hy_d, sizeof(float)*Nx*(Ny+1) );

  hipMemcpy(Ez_d, Ez, sizeof(float)*(Nx+1)*(Ny+1), hipMemcpyHostToDevice);
  hipMemcpy(Hx_d, Hx, sizeof(float)*(Nx+1)*Ny,     hipMemcpyHostToDevice);
  hipMemcpy(Hy_d, Hy, sizeof(float)*Nx*(Ny+1),     hipMemcpyHostToDevice);

  dim3 Dg(10,10,1), Db(10,10,1);

  for(int n = 0; n < NT; n++){

    update_Ez <<<Dg, Db>>> (Ez_d, Hx_d, Hy_d, Nx, Ny, CEz1, CEz2);
    add_Jz <<<Dg, Db>>> (Ez_d, Nx/2, Ny/2, (n-0.5)*Dt, Dt, EPS0, sig, 5.0*sig);
    hipDeviceSynchronize();

    update_Hx <<<Dg, Db>>> (Hx_d, Ez_d, Nx, Ny, CHx);
    update_Hy <<<Dg, Db>>> (Hy_d, Ez_d, Nx, Ny, CHy);

    hipMemcpy(Ez, Ez_d, sizeof(float)*(Nx+1)*(Ny+1), hipMemcpyDeviceToHost);
    std::string filename("ez_" + std::to_string(n) + ".dat");
    std::ofstream ofs(filename.c_str());
    for(int i = 0; i <= Nx; i++){
      for(int j = 0; j <= Ny; j++){
        ofs << i << " " << j << " "
        << Ez[idx_ez(i,j)] << "\n";
      }
      ofs << "\n";
    }
    ofs.close();

    hipDeviceSynchronize();
  }

  std::cout << Ez[idx_ez(25, 25)] << std::endl;

  hipFree(Ez_d);
  hipFree(Hx_d);
  hipFree(Hy_d);
}
