#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES
#include <cmath>
#include <complex>
#include <Eigen/Core>
#include <Eigen/Dense>

#include "fdtd3d.h"

void set_matrix(
    std::complex <float> zj, float *Cmat, float *Fmat,
    float *Nh, float *Ny ){
    
    float omg_c = E_Q*B_abs/E_M;

    for(int ir = Nr - ion_L; ir < Nr; ir++ ){
        int i = ir - (Nr - ion_L);

        for( int j = 0; j < Ntheta; j++ ){
            for( int k = 0; k < Nphi; k++ ){
                float omg_p = E_Q*std::sqrt( Nh[i*(Ntheta*Nphi) + j*Nphi + k]/E_M/EPS0 );
                std::complex <float> omg = omega -zj * Ny[i];
                std::complex <float> diag_comp = omg/(omg_c*omg_c - omg*omg);
                std::complex <float> offd_comp = zj * omg_c / (omg_c*omg_c - omg*omg);
                std::complex <float> coef = zj * EPS0 * omg_p*omg_p;

                Eigen::Matrix3d Sigma = Eigen::Matrix3d::Zero(3, 3);
                Sigma(0, 0) = real( coef*diag_comp );
                Sigma(1, 1) = real( coef*diag_comp );
                Sigma(0, 1) = real( coef*offd_comp );
                Sigma(1, 0) = real( -1.0*coef*offd_comp );
                Sigma(2, 2) = real( -1.0*coef/omg );

                Eigen::Matrix3d A =
                                EPS0/Dt*Eigen::Matrix3d::Identity(3, 3) + 0.5*Sigma;
                Eigen::Matrix3d B =
                                EPS0/Dt*Eigen::Matrix3d::Identity(3, 3) - 0.5*Sigma;
                Eigen::Matrix3d C = A.inverse()*B;
                Eigen::Matrix3d F = 1.0/Dt*A.inverse();

                for( int m = 0; m < 3; m++ ){
                    for( int n = 0; n < 3; n++ ){
                        int idx = i*(Ntheta*Nphi*3*3) + j*(Nphi*3*3) + k*(3*3)
                                            + m*3 + n;
                        Cmat[ idx ] = C(m, n);
                        Fmat[ idx ] = F(m, n);
                    }
                }

            }
        }

    }
}