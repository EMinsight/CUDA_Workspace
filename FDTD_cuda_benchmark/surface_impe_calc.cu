#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES
#include <cmath>
#include <complex>
#include "fdtd3d.h"

std::complex <float> surface_impe( std::complex <float> zj )
{
    float conduct = SIGMA_VERY_DRY_GROUND;

    std::complex <float> z = Z0/std::sqrt(EPSR - (zj*conduct/EPS0/omega) );

    return z;
}