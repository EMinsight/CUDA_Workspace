#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES
#include <cmath>
#include "main.h"

__global__ void update_Hy( int Nx, int Ny, float* Hy_d, float* Ez_d, float CHy )
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;

    if( (i >= 0) && (i < Nx) && (j > 0) && (j < Ny) ){
        Hy_d[i + j*Nx] = Hy_d[i + j*Nx]
                + CHy*( Ez_d[(i + 1) + j*(Nx+1)] - Ez_d[i + j*(Nx+1)] );
    }

}