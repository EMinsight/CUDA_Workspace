
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

int main( void ){

    int nElem = 1024;

    dim3 block(1024);
    dim3 grid( ( nElem + block.x - 1 ) / block.x );
    std::cout << "grid.x : " << grid.x << "  block.x : " << block.x << std::endl;

    // ブロックをリセット //
    block.x = 512;
    grid.x = (nElem + block.x - 1) / block.x;
    std::cout << "grid.x : " << grid.x << "  block.x : " << block.x << std::endl;

     // ブロックをリセット //
    block.x = 256;
    grid.x = (nElem + block.x - 1) / block.x;
    std::cout << "grid.x : " << grid.x << "  block.x : " << block.x << std::endl;

     // ブロックをリセット //
    block.x = 128;
    grid.x = (nElem + block.x - 1) / block.x;
    std::cout << "grid.x : " << grid.x << "  block.x : " << block.x << std::endl;

    // デバイスをリセット //
    hipDeviceReset();

    return(0);

}