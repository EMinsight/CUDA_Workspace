#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES
#include <cmath>

#include "main.h"

__global__ void add_Jz( int i_s, int j_s, float* Ez_d, float t, float Dt, float t0, float sig )
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;

    if( (i == i_s)&&(j == j_s) ){
        Ez_d[ i + j*( Nx + 1 ) ] = Ez_d[ i + j*( Nx + 1 ) ]
            -   Dt/EPS0 * std::exp( -(t - t0)*(t - t0) / 2.0 / sig / sig );
    }

}