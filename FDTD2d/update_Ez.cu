#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES
#include <cmath>

#include "main.h"

__global__ void update_Ez( int Nx, int Ny, float* Ez_d, float* Hx_d, float* Hy_d, float CEz1, float CEz2 )
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;

    if( (i > 0) && (i < Nx) && ( j > 0) && (j < Ny) ){
        Ez_d[ i + j*(Nx + 1) ] = Ez_d[ i + j*(Nx + 1) ] + CEz1*( Hy_d[ i + j*Nx ] - Hy_d[ i - 1 + j*Nx ] )
                                                - CEz2*( Hx_d[ i + j*(Nx+1) ] - Hx_d[i + (j-1)*(Nx+1)] );
    }

}