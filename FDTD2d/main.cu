#define _USE_MATH_DEFINES

#include <iostream>
#include <cmath>
#include <fstream>
#include <string>
#include <stdio.h>

#include "main.h"

int main( void )
{
    /* Memory allocate (host) */
    float *Ez = new float[ ( Nx+1 )*( Ny+1 )];
    float *Hx = new float[ ( Nx+1 )*Ny ];
    float *Hy = new float[ Nx*( Ny+1 ) ];

    /* initialize */
    for( int i = 0; i < Nx+1; i++ ){
        for( int j = 0; j < Ny+1; j++ ) Ez[ idx_Ez(i, j) ] = 0.0;
    }

    for( int i = 0; i < Nx+1; i++ ){
        for( int j = 0; j < Ny; j++ ) Hx[ idx_Hx(i, j) ] = 0.0;
    }

    for( int i = 0; i < Nx; i++ ){
        for( int j = 0; j < Ny+1; j++ ) Hy[ idx_Hy(i, j) ] = 0.0;
    }

    /* Memory allocate (device) */
    float *Ez_d, *Hx_d, *Hy_d;
    hipMalloc( (void**)&Ez_d, sizeof(float)*(Nx+1)*(Ny+1) );
    hipMalloc( (void**)&Hx_d, sizeof(float)*(Nx+1)*Ny );
    hipMalloc( (void**)&Hy_d, sizeof(float)*Nx*(Ny+1) );

    /* Copy to Device */
    hipMemcpy( Ez_d, Ez, sizeof(float)*(Nx+1)*(Ny+1),  hipMemcpyHostToDevice );
    hipMemcpy( Hx_d, Hx, sizeof(float)*(Nx+1)*Ny,  hipMemcpyHostToDevice );
    hipMemcpy( Hy_d, Hy, sizeof(float)*Nx*(Ny+1),  hipMemcpyHostToDevice );

    dim3 Dg(10,10,1), Db(10,10,1);

    int NT{ int(Tmax/Dt) };

    const float CEz1 { Dt/EPS0/Dx };
    const float CEz2 { Dt/EPS0/Dy };
    const float CHx { Dt/MU0/Dy };
    const float CHy { Dt/MU0/Dx };

    for( int n = 0; n < NT; n++ ){
        
        float t { float(((float)n-0.5)*Dt) };

        add_Jz <<<Dg, Db>>> ( i_s, j_s, Ez_d, t ,Dt, t0, sig );
        update_Ez <<<Dg, Db>>> ( Nx, Ny, Ez_d, Hx_d, Hy_d, CEz1, CEz2 );

        /* Cuda 同期 */
        hipDeviceSynchronize();

        update_Hx <<<Dg, Db>>> ( Nx, Ny, Hx_d, Ez_d, CHx );
        update_Hy <<<Dg, Db>>> ( Nx, Ny, Hy_d, Ez_d, CHy );

        hipMemcpy( Ez, Ez_d, sizeof(float)*(Nx+1)*(Ny+1), hipMemcpyDeviceToHost );
        std::string filename = "./result/ez_" + std::to_string(n) + ".dat";
        std::ofstream ofs(filename.c_str());
        for( int i = 0; i <= Nx; i++ ){
            for( int j = 0; j <= Ny; j++ ){
                ofs << i << " " << j << " " << Ez[idx_Ez(i, j)] << "\n";
            }
            ofs << "\n";
        }
        ofs.close();

        hipDeviceSynchronize();

    }

    hipFree( Ez_d );
    hipFree( Hx_d );
    hipFree( Hy_d );

}