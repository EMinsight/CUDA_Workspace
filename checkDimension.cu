#include <hip/hip_runtime.h>
#include <stdio.h>

#include <iostream>

__global__ void checkIndex( void ){
    /*std::cout << "threadIdx : " << threadIdx.x << " " << threadIdx.y << " " << threadIdx.z << std::endl
    << "blockIdx : " << blockIdx.x << " " << blockIdx.y << " " << blockIdx.z << std::endl
    << "blockDim : " << blockDim.x << " " << blockDim.y << " " << blockDim.z << std::endl
    << "gridDim : " << gridDim.x << " " << gridDim.y << " " << gridDim.z << std::endl;*/
    printf("threadIdx: (%d, %d, %d) blockIdx: (%d, %d, %d) blockDim: (%d, %d, %d) "
    "gridDim: (%d, %d, %d)\n", threadIdx.x, threadIdx.y, threadIdx.z,
                                blockIdx.z, blockIdx.y, blockIdx.z,
                                blockDim.x, blockDim.y, blockDim.z,
                                gridDim.x, gridDim.y, gridDim.z );
}

int main( int argc, char** argv ){

    int nElem( 6 );

    dim3 block(3);
    dim3 grid( ( nElem + block.x - 1) / block.x );

    std::cout << "grid_x : " << grid.x << " grid_y : " << grid.y << " grid_z : " << grid.z << std::endl;
    std::cout << "block_x : " << block.x << " block_y : " << block.y << " block_z : " << block.z << std::endl;

    checkIndex<<<grid, block>>> ();

    hipDeviceReset();

    return 0;
}