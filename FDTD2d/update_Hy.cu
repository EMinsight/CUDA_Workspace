#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES
#include <cmath>
#include "main.h"

__global__ void update_Hy( int Nx, int Ny, float* Hy, float* Ez, float CHy )
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;

    if( (i >= 0) && (i < Nx) && (j > 0) && (j < Ny) ){
        Hy[i + j*Nx] = Hy[i + j*Nx]
                        + CHy*( Ez[i + 1 + j*(Nx+1)] - Ez[i + j*(Nx+1)] );
    }

}