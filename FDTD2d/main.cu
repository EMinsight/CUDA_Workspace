#define _USE_MATH_DEFINES

#include <iostream>
#include <cmath>
#include <fstream>
#include <string>
#include <stdio.h>



int main( void )
{
    /* Memory allocate (host) */
    float *Ez = new float[ ( Nx+1 )*( Ny+1 )];
    float *Hx = new float[ ( Nx+1 )*Ny ];
    float *Hy = new float[ Nx*( Ny+1 ) ];

    /* initialize */
    for( int i = 0; i < Nx+1; i++ ){
        for( int j = 0; j < Ny+1; j++ ) Ez[ idx_Ez(i, j) ] = 0.0;
    }

    for( int i = 0; i < Nx+1; i++ ){
        for( int j = 0; j < Ny; j++ ) Hx[ idx_Hx(i, j) ] = 0.0;
    }

    for( int i = 0; i < Nx; i++ ){
        for( int j = 0; j < Ny+1; j++ ) Hy[ idx_Hy(i, j) ] = 0.0;
    }

    /* Memory allocate (device) */
    /*float *Ez_d, *Hx_d, *Hy_d;
    hipMalloc( (void**)&Ez_d, sizeof(float)*(Nx+1)*(Ny+1) );
    hipMalloc( (void**)&Hx_d, sizeof(float)*(Nx+1)*Ny );
    hipMalloc( (void**)&Hy_d, sizeof(float)*Nx*(Ny+1) );*/

    /* Copy to Device */
    /*hipMemcpy( Ez_d, Ez, sizeof(float)*(Nx+1)*(Ny+1),  cudaMemcpyToDevice );
    hipMemcpy( Hx_d, Hx, sizeof(float)*(Nx+1)*Ny,  cudaMemcpyToDevice );
    hipMemcpy( Hy_d, Hy, sizeof(float)*Nx*(Ny+1),  cudaMemcpyToDevice );*/

    /* a + b = c */
    float *a = new float[ Nx ];
    float *b = new float[ Nx ];
    float *c = new float[ Nx ];

    for( int i = 0; i < Nx; i++ ){
        a[i] = (float)i;
        b[i] = 2.5*(float)i;
        c[i] = 0.0;
    }

    // memory allocate (device) //
    float *a_d, *b_d, *c_d;
    hipMalloc( (void**)&a_d, sizeof(float)*Nx );
    hipMalloc( (void**)&b_d, sizeof(float)*Nx );
    hipMalloc( (void**)&c_d, sizeof(float)*Nx );

    // Copy to Device //
    hipMemcpy( a_d, a, sizeof(float)*Nx,   cudaMemcpyToDevice );
    hipMemcpy( b_d, b, sizeof(float)*Nx,   cudaMemcpyToDevice );
    hipMemcpy( c_d, c, sizeof(float)*Nx,   cudaMemcpyToDevice );

    dim3 Dg(10,10,1), Db(10,10,1);

    add <<<Dg, Db>>> ( a, b, c );

    //int NT{ Tmax/Dt };

    /*for( int n = 0; n < NT; n++ ){

    }*/

}