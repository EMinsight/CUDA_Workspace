#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES
#include <iostream>
#include <cmath>
#include <complex>
#include <fstream>
#include <string>
#include <chrono>

#include "fdtd3d.h"

const float R_r{ 100.0e3 };

const float delta_r = R_r/float(Nr);
const float delta_theta = 1.0e3/float(R0);
const float delta_phi = 1.0e3/float(R0);
const float Dt = float( 0.99/C0/std::sqrt(1.0/delta_r/delta_r
 + 1.0/R0/R0/delta_theta/delta_theta
 + 1.0/R0/R0/std::sin(THETA0)/std::sin(THETA0)/delta_phi/delta_phi) );
const float inv_Dt = 1.0/Dt;
const float sigma_t = 7*Dt;
const float t0 = 6.0*sigma_t;

// PML info //
const int L { 10 };
const float M {3.5 };
const float R { 1.0e-6 };

const float sigma_th_max = float( -(M + 1.0)*C0*std::log(R)/2.0/double(L)/delta_theta/R0 );
const float sigma_phi_max = float( -(M + 1.0)*C0*std::log(R)/2.0/double(L)/delta_phi/R0 );

// Ionosphere info //
constexpr float Alt_lower_ionosphere { 60.0e3 };
const int ion_L = int( (R_r - Alt_lower_ionosphere )/delta_r );
const float freq { 22.2e3 };
const float omega = 2.0*M_PI*freq;

// Geomagnetic info //
const float B_abs = float( 4.6468e-5 );
const float Dec = float( -7.0*M_PI/180.0 );
const float Inc = float( 49.0*M_PI/180.0 );
const float Azim = float( 61.0*M_PI/180.0 );

void fdtd_calc( )
{
    int t_step = 1800;
    float t;
    float J;
    int NEW;
    int OLD;
    std::complex <float> zj(0.0, 1.0);

    float* Hr, *Htheta, *Hphi;
    Hr = new float [ (Nr + 1)*Ntheta*(Nphi) ];
    Htheta = new float [ Nr*(Ntheta+1)*Nphi ];
    Hphi = new float [Nr*Ntheta*(Nphi+1) ];



}